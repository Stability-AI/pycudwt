#include "hip/hip_runtime.h"
/// ****************************************************************************
/// ***************** Common utilities and  CUDA Kernels  **********************
/// ****************************************************************************

//~ #include "utils.h"
#include "common.h"
#define W_SIGN(a) ((a > 0) ? (1.0f) : (-1.0f))
#define SQRT_2 1.4142135623730951
#include <hipblas.h>


/// soft thresholding of the detail coefficients (2D)
/// Must be lanched with block size (Nc, Nr) : the size of the current coefficient vector
__global__ void w_kern_soft_thresh(float* c_h, float* c_v, float* c_d, float beta, int Nr, int Nc) {
    int gidx = threadIdx.x + blockIdx.x*blockDim.x;
    int gidy = threadIdx.y + blockIdx.y*blockDim.y;
    float val = 0.0f;
    if (gidx < Nc && gidy < Nr) {
        val = c_h[gidy*Nc + gidx];
        c_h[gidy*Nc + gidx] = copysignf(max(fabsf(val)-beta, 0.0f), val);

        val = c_v[gidy*Nc + gidx];
        c_v[gidy*Nc + gidx] = copysignf(max(fabsf(val)-beta, 0.0f), val);

        val = c_d[gidy*Nc + gidx];
        c_d[gidy*Nc + gidx] = copysignf(max(fabsf(val)-beta, 0.0f), val);
    }
}

/// soft thresholding of the detail coefficients (1D)
/// Must be lanched with block size (Nc, Nr) : the size of the current coefficient vector
// CHECKME: consider merging this kernel with the previous kernel
__global__ void w_kern_soft_thresh_1d(float* c_d, float beta, int Nr, int Nc) {
    int gidx = threadIdx.x + blockIdx.x*blockDim.x;
    int gidy = threadIdx.y + blockIdx.y*blockDim.y;
    float val = 0.0f;
    if (gidx < Nc && gidy < Nr) {
        val = c_d[gidy*Nc + gidx];
        c_d[gidy*Nc + gidx] = copysignf(max(fabsf(val)-beta, 0.0f), val);
    }
}

/// soft thresholding of the approximation coefficients (2D and 1D)
/// Must be lanched with block size (Nc, Nr) : the size of the current coefficient vector
__global__ void w_kern_soft_thresh_appcoeffs(float* c_a, float beta, int Nr, int Nc) {
    int gidx = threadIdx.x + blockIdx.x*blockDim.x;
    int gidy = threadIdx.y + blockIdx.y*blockDim.y;
    float val = 0.0f;
    if (gidx < Nc && gidy < Nr) {
        val = c_a[gidy*Nc + gidx];
        c_a[gidy*Nc + gidx] = copysignf(max(fabsf(val)-beta, 0.0f), val);
    }
}



/// Hard thresholding of the detail coefficients (2D)
/// Must be lanched with block size (Nc, Nr) : the size of the current coefficient vector
__global__ void w_kern_hard_thresh(float* c_h, float* c_v, float* c_d, float beta, int Nr, int Nc) {
    int gidx = threadIdx.x + blockIdx.x*blockDim.x;
    int gidy = threadIdx.y + blockIdx.y*blockDim.y;
    float val = 0.0f;
    if (gidx < Nc && gidy < Nr) {
        val = c_h[gidy*Nc + gidx];
        c_h[gidy*Nc + gidx] = max(W_SIGN(fabsf(val)-beta), 0.0f)*val;

        val = c_v[gidy*Nc + gidx];
        c_v[gidy*Nc + gidx] = max(W_SIGN(fabsf(val)-beta), 0.0f)*val;

        val = c_d[gidy*Nc + gidx];
        c_d[gidy*Nc + gidx] = max(W_SIGN(fabsf(val)-beta), 0.0f)*val;
    }
}


/// Hard thresholding of the detail coefficients (1D)
/// Must be lanched with block size (Nc, Nr) : the size of the current coefficient vector
// CHECKME: consider merging this kernel with the previous kernel
__global__ void w_kern_hard_thresh_1d(float* c_d, float beta, int Nr, int Nc) {
    int gidx = threadIdx.x + blockIdx.x*blockDim.x;
    int gidy = threadIdx.y + blockIdx.y*blockDim.y;
    float val = 0.0f;
    if (gidx < Nc && gidy < Nr) {
        val = c_d[gidy*Nc + gidx];
        c_d[gidy*Nc + gidx] = max(W_SIGN(fabsf(val)-beta), 0.0f)*val;
    }
}


/// Hard thresholding of the approximation coefficients (2D and 1D)
__global__ void w_kern_hard_thresh_appcoeffs(float* c_a, float beta, int Nr, int Nc) {
    int gidx = threadIdx.x + blockIdx.x*blockDim.x;
    int gidy = threadIdx.y + blockIdx.y*blockDim.y;
    float val = 0.0f;
    if (gidx < Nc && gidy < Nr) {
        val = c_a[gidy*Nc + gidx];
        c_a[gidy*Nc + gidx] = max(W_SIGN(fabsf(val)-beta), 0.0f)*val;
    }
}

/// Circular shift of the image (2D and 1D)
__global__ void w_kern_circshift(float* d_image, float* d_out, int Nr, int Nc, int sr, int sc) {
    int gidx = threadIdx.x + blockIdx.x*blockDim.x;
    int gidy = threadIdx.y + blockIdx.y*blockDim.y;
    if (gidx < Nc && gidy < Nr) {
        int r = gidy - sr, c = gidx - sc;
        if (r < 0) r += Nr;
        if (c < 0) c += Nc;
        d_out[gidy*Nc + gidx] = d_image[r*Nc + c];
    }
}



/// ****************************************************************************
/// ******************** Common CUDA Kernels calls *****************************
/// ****************************************************************************

void w_call_soft_thresh(float** d_coeffs, float beta, w_info winfos, int do_thresh_appcoeffs, int normalize, int threshold_cousins) {
    int tpb = 16; // Threads per block
    dim3 n_threads_per_block = dim3(tpb, tpb, 1);
    dim3 n_blocks;
    int Nr = winfos.Nr, Nc = winfos.Nc, do_swt = winfos.do_swt, nlevels = winfos.nlevels, ndims = winfos.ndims;
    int Nr2 = Nr, Nc2 = Nc;
    if (!do_swt) {
        if (ndims > 1) Nr2 /= 2;
        Nc2 /= 2;
    }
    if (do_thresh_appcoeffs) {
        float beta2 = beta;
        if (normalize > 0) { // beta2 = beta/sqrt(2)^nlevels
            int nlevels2 = nlevels/2;
            beta2 /= (1 << nlevels2);
            if (nlevels2 *2 != nlevels) beta2 /= SQRT_2;
        }
        n_blocks = dim3(w_iDivUp(Nc2, tpb), w_iDivUp(Nr2, tpb), 1);
        w_kern_soft_thresh_appcoeffs<<<n_blocks, n_threads_per_block>>>(d_coeffs[0], beta2, Nr2, Nc2);
    }
    for (int i = 0; i < nlevels; i++) {
        if (!do_swt) {
            if (ndims > 1) Nr /= 2;
            Nc /= 2;
        }
        if (normalize > 0) beta /= SQRT_2;
        n_blocks = dim3(w_iDivUp(Nc, tpb), w_iDivUp(Nr, tpb), 1);
        if (ndims > 1) w_kern_soft_thresh<<<n_blocks, n_threads_per_block>>>(d_coeffs[3*i+1], d_coeffs[3*i+2], d_coeffs[3*i+3], beta, Nr, Nc);
        else w_kern_soft_thresh_1d<<<n_blocks, n_threads_per_block>>>(d_coeffs[i+1], beta, Nr, Nc);
    }
}


void w_call_hard_thresh(float** d_coeffs, float beta, w_info winfos, int do_thresh_appcoeffs, int normalize) {
    int tpb = 16; // Threads per block
    dim3 n_threads_per_block = dim3(tpb, tpb, 1);
    dim3 n_blocks;
    int Nr = winfos.Nr, Nc = winfos.Nc, do_swt = winfos.do_swt, nlevels = winfos.nlevels, ndims = winfos.ndims;
    int Nr2 = Nr, Nc2 = Nc;
    if (!do_swt) {
        if (ndims > 1) Nr2 /= 2;
        Nc2 /= 2;
    }
    float beta2 = beta;
    if (do_thresh_appcoeffs) {
        if (normalize > 0) { // beta2 = beta/sqrt(2)^nlevels
            int nlevels2 = nlevels/2;
            beta2 /= (1 << nlevels2);
            if (nlevels2 *2 != nlevels) beta2 /= SQRT_2;
        }
        n_blocks = dim3(w_iDivUp(Nc2, tpb), w_iDivUp(Nr2, tpb), 1);
        w_kern_hard_thresh_appcoeffs<<<n_blocks, n_threads_per_block>>>(d_coeffs[0], beta, Nr2, Nc2);
    }
    for (int i = 0; i < nlevels; i++) {
        if (!do_swt) {
            if (ndims > 1) Nr /= 2;
            Nc /= 2;
        }
        if (normalize > 0) beta /= SQRT_2;
        n_blocks = dim3(w_iDivUp(Nc, tpb), w_iDivUp(Nr, tpb), 1);
        if (ndims > 1) w_kern_hard_thresh<<<n_blocks, n_threads_per_block>>>(d_coeffs[3*i+1], d_coeffs[3*i+2], d_coeffs[3*i+3], beta, Nr, Nc);
        else w_kern_hard_thresh_1d<<<n_blocks, n_threads_per_block>>>(d_coeffs[i+1], beta, Nr, Nc);
    }
}


void w_shrink(float** d_coeffs, float beta, w_info winfos, int do_thresh_appcoeffs) {
    int Nr = winfos.Nr, Nc = winfos.Nc, do_swt = winfos.do_swt, nlevels = winfos.nlevels, ndims = winfos.ndims;
    int Nr2 = Nr, Nc2 = Nc;
    if (!do_swt) {
        if (ndims > 1) Nr2 /= 2;
        Nc2 /= 2;
    }
    if (do_thresh_appcoeffs) {
        hipblasSscal(Nr2*Nc2, 1.0f/(1.0f + beta), d_coeffs[0], 1);
    }
    for (int i = 0; i < nlevels; i++) {
        if (!do_swt) {
            if (ndims > 1) Nr /= 2;
            Nc /= 2;
        }
        if (ndims == 2) {
            hipblasSscal(Nr*Nc, 1.0f/(1.0f + beta), d_coeffs[3*i+1], 1);
            hipblasSscal(Nr*Nc, 1.0f/(1.0f + beta), d_coeffs[3*i+2], 1);
            hipblasSscal(Nr*Nc, 1.0f/(1.0f + beta), d_coeffs[3*i+3], 1);
        }
        else { // 1D
            hipblasSscal(Nr*Nc, 1.0f/(1.0f + beta), d_coeffs[i+1], 1);
        }
    }
}





// if inplace = 1, the result is in "d_image" ; otherwise result is in "d_image2".
void w_call_circshift(float* d_image, float* d_image2, w_info winfos, int sr, int sc, int inplace) {
    int Nr = winfos.Nr, Nc = winfos.Nc, ndims = winfos.ndims;
    // Modulus in C can be negative
    if (sr < 0) sr += Nr; // or do while loops to ensure positive numbers
    if (sc < 0) sc += Nc;
    int tpb = 16; // Threads per block
    sr = sr % Nr;
    sc = sc % Nc;
    if (ndims == 1) sr = 0;
    dim3 n_blocks = dim3(w_iDivUp(Nc, tpb), w_iDivUp(Nr, tpb), 1);
    dim3 n_threads_per_block = dim3(tpb, tpb, 1);
    if (inplace) {
        hipMemcpy(d_image2, d_image, Nr*Nc*sizeof(float), hipMemcpyDeviceToDevice);
        w_kern_circshift<<<n_blocks, n_threads_per_block>>>(d_image2, d_image, Nr, Nc, sr, sc);
    }
    else {
        w_kern_circshift<<<n_blocks, n_threads_per_block>>>(d_image, d_image2, Nr, Nc, sr, sc);
    }
}


/// Creates an allocated/padded device array : [ An, H1, V1, D1, ..., Hn, Vn, Dn]
float** w_create_coeffs_buffer(w_info winfos) {
    int Nr = winfos.Nr, Nc = winfos.Nc, do_swt = winfos.do_swt, nlevels = winfos.nlevels;
    int Nr0 = Nr, Nc0 = Nc;
    if (!do_swt) { Nr0 /= 2; Nc0 /= 2; }
    float** res = (float**) calloc(3*nlevels+1, sizeof(float*));
    // Coeffs (H, V, D)
    for (int i = 1; i < 3*nlevels+1; i += 3) {
        if (!do_swt) {
            Nr /= 2;
            Nc /= 2;
        }
        hipMalloc(&(res[i]), Nr*Nc*sizeof(float));
        hipMemset(res[i], 0, Nr*Nc*sizeof(float));
        hipMalloc(&(res[i+1]), Nr*Nc*sizeof(float));
        hipMemset(res[i+1], 0, Nr*Nc*sizeof(float));
        hipMalloc(&(res[i+2]), Nr*Nc*sizeof(float));
        hipMemset(res[i+2], 0, Nr*Nc*sizeof(float));
    }
    // App coeff (last scale). They are also useful as a temp. buffer for the reconstruction, hence a bigger size
    hipMalloc(&(res[0]), Nr0*Nc0*sizeof(float));
    hipMemset(res[0], 0, Nr0*Nc0*sizeof(float));

    return res;
}


/// Creates an allocated/padded device array : [ An, D1, ..., Dn]
float** w_create_coeffs_buffer_1d(w_info winfos) {
    int Nr = winfos.Nr, Nc = winfos.Nc, do_swt = winfos.do_swt, nlevels = winfos.nlevels;
    int Nc0 = Nc;
    if (!do_swt) Nc0 /= 2;
    float** res = (float**) calloc(nlevels+1, sizeof(float*));
    // Det coeffs
    for (int i = 1; i < nlevels+1; i++) {
        if (!do_swt) Nc /= 2;
        hipMalloc(&(res[i]), Nr*Nc*sizeof(float));
        hipMemset(res[i], 0, Nr*Nc*sizeof(float));
    }
    // App coeff (last scale). They are also useful as a temp. buffer for the reconstruction, hence a bigger size
    hipMalloc(&(res[0]), Nr*Nc0*sizeof(float));
    hipMemset(res[0], 0, Nr*Nc0*sizeof(float));
    return res;
}



/// Deep free of wavelet coefficients
void w_free_coeffs_buffer(float** coeffs, int nlevels) {
    for (int i = 0; i < 3*nlevels+1; i++) hipFree(coeffs[i]);
    free(coeffs);
}

void w_free_coeffs_buffer_1d(float** coeffs, int nlevels) {
    for (int i = 0; i < nlevels+1; i++) hipFree(coeffs[i]);
    free(coeffs);
}


/// Deep copy of wavelet coefficients. All structures must be allocated.
void w_copy_coeffs_buffer(float** dst, float** src, w_info winfos) {
    int Nr = winfos.Nr, Nc = winfos.Nc, nlevels = winfos.nlevels, do_swt = winfos.do_swt;
    // Coeffs (H, V, D)
    for (int i = 1; i < 3*nlevels+1; i += 3) {
        if (!do_swt) {
            Nr /= 2;
            Nc /= 2;
        }
        hipMemcpy(dst[i], src[i], Nr*Nc*sizeof(float), hipMemcpyDeviceToDevice);
        hipMemcpy(dst[i+1], src[i+1], Nr*Nc*sizeof(float), hipMemcpyDeviceToDevice);
        hipMemcpy(dst[i+2], src[i+2], Nr*Nc*sizeof(float), hipMemcpyDeviceToDevice);
    }
    // App coeff (last scale)
    hipMemcpy(dst[0], src[0], Nr*Nc*sizeof(float), hipMemcpyDeviceToDevice);
}


void w_copy_coeffs_buffer_1d(float** dst, float** src, w_info winfos) {
    int Nr = winfos.Nr, Nc = winfos.Nc, nlevels = winfos.nlevels, do_swt = winfos.do_swt;
    // Det Coeffs
    for (int i = 1; i < nlevels+1; i++) {
        if (!do_swt) Nc /= 2;
        hipMemcpy(dst[i], src[i], Nr*Nc*sizeof(float), hipMemcpyDeviceToDevice);
    }
    // App coeff (last scale)
    hipMemcpy(dst[0], src[0], Nr*Nc*sizeof(float), hipMemcpyDeviceToDevice);
}



///
/// ----------------------------------------------------------------------------
///




void w_add_coeffs(float** dst, float** src, w_info winfos, float alpha) {
    int Nr = winfos.Nr, Nc = winfos.Nc, do_swt = winfos.do_swt, nlevels = winfos.nlevels;
    // Coeffs (H, V, D)
    for (int i = 1; i < 3*nlevels+1; i += 3) {
        if (!do_swt) {
            Nr /= 2;
            Nc /= 2;
        }
        hipblasSaxpy(Nr*Nc, alpha, src[i], 1, dst[i], 1);
        hipblasSaxpy(Nr*Nc, alpha, src[i+1], 1, dst[i+1], 1);
        hipblasSaxpy(Nr*Nc, alpha, src[i+2], 1, dst[i+2], 1);
    }
    // App coeff (last scale)
    hipblasSaxpy(Nr*Nc, alpha, src[0], 1, dst[0], 1);
}


void w_add_coeffs_1d(float** dst, float** src, w_info winfos, float alpha) {
    int Nr = winfos.Nr, Nc = winfos.Nc, do_swt = winfos.do_swt, nlevels = winfos.nlevels;
    // Det Coeffs
    for (int i = 1; i < nlevels+1; i++) {
        if (!do_swt) Nc /= 2;
        hipblasSaxpy(Nr*Nc, alpha, src[i], 1, dst[i], 1);
    }
    // App coeff (last scale)
    hipblasSaxpy(Nr*Nc, alpha, src[0], 1, dst[0], 1);
}



